#include "hip/hip_runtime.h"
/*
	Optimized keccak256_gpu_hash_ZP CUDA implementation for CCMINER written by (c) sp in january 2021
	For the ZenProtocol coin
*/

#include "miner.h"
#ifdef __cplusplus
#include <cstdint>
#else
#include <stdint.h>
#endif
#include <memory.h>
#include "cuda_helper.h"


#define UINT2(x,y) make_uint2(x,y)

static uint32_t *d_KNonce[MAX_GPUS];

__constant__ uint32_t pTarget[8];
__constant__ uint64_t keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint2 keccak_round_constants35[24] = {
		{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
		{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
		{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
		{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
		{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
		{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
		{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
		{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
		{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
		{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};


__constant__ uint2 __align__(16) c_PaddedMessageZP_PRE[23];
__constant__ uint2 __align__(16) c_PaddedMessage80[10]; // padded message (80 bytes + padding?)
#define bitselect(a, b, c) ((a) ^ ((c) & ((b) ^ (a))))

static void __forceinline__ __device__ keccak_block(uint2 *s)
{
	uint2 bc[5], tmpxor[5], tmp1, tmp2;
//	uint2 s[25];

#pragma unroll 1
	for (int i= 0; i < 24; i++) 
	{
#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL8(s[19] ^ bc[3]);
		s[19] = ROR8(s[23] ^ bc[2]);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0] ^= keccak_round_constants35[i];
	}
}

__global__	__launch_bounds__(512)
void keccak256_gpu_hash_80(uint32_t threads, uint32_t startNounce,  uint32_t *const __restrict__ resNounce)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = startNounce + thread;
		uint2 bc[5], tmpxor[5], tmp1, tmp2;
		uint2 s[25];
		
		s[9] = make_uint2(c_PaddedMessage80[9].x, cuda_swab32(nounce));
		s[10] = make_uint2( 0x06, 0);
		s[16] = make_uint2(0, 0x80000000);

		tmpxor[0] = c_PaddedMessage80[0] ^ c_PaddedMessage80[5] ^ s[10];
		tmpxor[1] = c_PaddedMessage80[1] ^ c_PaddedMessage80[6] ^ s[16];
		tmpxor[2] = c_PaddedMessage80[2] ^ c_PaddedMessage80[7];
		tmpxor[3] = c_PaddedMessage80[3] ^ c_PaddedMessage80[8];
		tmpxor[4] = c_PaddedMessage80[4] ^ s[9];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = c_PaddedMessage80[1] ^ bc[0];

		s[0] = c_PaddedMessage80[0] ^ bc[4];
		s[1] = ROL2(c_PaddedMessage80[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(bc[1], 61);
		s[22] = ROL2(bc[3], 39);
		s[14] = ROL2(bc[4], 18);
		s[20] = ROL2(c_PaddedMessage80[2] ^ bc[1], 62);
		s[2] = ROL2(bc[1], 43);
		s[12] = ROL2(bc[2], 25);
		s[13] = ROL8(bc[3]);
		s[19] = ROR8(bc[2]);
		s[23] = ROL2(bc[4], 41);
		s[15] = ROL2(c_PaddedMessage80[4] ^ bc[3], 27);
		s[4] = ROL2(bc[3], 14);
		s[24] = ROL2(bc[0], 2);
		s[21] = ROL2(c_PaddedMessage80[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(c_PaddedMessage80[5] ^ bc[4], 36);
		s[5] = ROL2(c_PaddedMessage80[3] ^ bc[2], 28);
		s[3] = ROL2( bc[2], 21);
		s[18] = ROL2(bc[1], 15);
		s[17] = ROL2(bc[0], 10);
		s[11] = ROL2(c_PaddedMessage80[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0].x ^= 1;

#pragma unroll 2
		for (int i = 1; i < 23; i++) 
		{

#pragma unroll
			for (uint32_t x = 0; x < 5; x++)
				tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

			bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
			bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
			bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
			bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
			bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

			tmp1 = s[1] ^ bc[0];

			s[0] ^= bc[4];
			s[1] = ROL2(s[6] ^ bc[0], 44);
			s[6] = ROL2(s[9] ^ bc[3], 20);
			s[9] = ROL2(s[22] ^ bc[1], 61);
			s[22] = ROL2(s[14] ^ bc[3], 39);
			s[14] = ROL2(s[20] ^ bc[4], 18);
			s[20] = ROL2(s[2] ^ bc[1], 62);
			s[2] = ROL2(s[12] ^ bc[1], 43);
			s[12] = ROL2(s[13] ^ bc[2], 25);
			s[13] = ROL8(s[19] ^ bc[3]);
			s[19] = ROR8(s[23] ^ bc[2]);
			s[23] = ROL2(s[15] ^ bc[4], 41);
			s[15] = ROL2(s[4] ^ bc[3], 27);
			s[4] = ROL2(s[24] ^ bc[3], 14);
			s[24] = ROL2(s[21] ^ bc[0], 2);
			s[21] = ROL2(s[8] ^ bc[2], 55);
			s[8] = ROL2(s[16] ^ bc[0], 45);
			s[16] = ROL2(s[5] ^ bc[4], 36);
			s[5] = ROL2(s[3] ^ bc[2], 28);
			s[3] = ROL2(s[18] ^ bc[2], 21);
			s[18] = ROL2(s[17] ^ bc[1], 15);
			s[17] = ROL2(s[11] ^ bc[0], 10);
			s[11] = ROL2(s[7] ^ bc[1], 6);
			s[7] = ROL2(s[10] ^ bc[4], 3);
			s[10] = ROL2(tmp1, 1);

			tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
			tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
			tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
			tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
			tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
			s[0] ^= keccak_round_constants35[i];
		}
		uint2 t[5];
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		s[0] ^= t[4] ^ ROL2(t[1], 1);
		s[18] ^= t[2] ^ ROL2(t[4], 1);
		s[24] ^= t[3] ^ ROL2(t[0], 1);

		s[3] = ROL2(s[18], 21) ^ ((~ROL2(s[24], 14)) & s[0]);


		if (devectorize(s[3]) <= ((uint64_t*)pTarget)[3])
		{
			uint32_t tmp = atomicCAS(resNounce, 0xffffffff, nounce);
			if (tmp != 0xffffffff)
				resNounce[1] = nounce;
		}
	}
}

__device__ __forceinline__
uint64_t xor5(uint64_t a, uint64_t b, uint64_t c, uint64_t d, uint64_t e)
{
	uint64_t result;
	asm("xor.b64 %0, %1, %2;" : "=l"(result) : "l"(d), "l"(e));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(c));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(b));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(a));
	return result;
}

__device__ __forceinline__
uint2 xor3x2(const uint2 a, const uint2 b, const uint2 c)
{
	uint2 result; // = result = a^b^c;
	asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.x) : "r"(a.x), "r"(b.x), "r"(c.x)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
	asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(result.y) : "r"(a.y), "r"(b.y), "r"(c.y)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
	return result;
}

__device__ __forceinline__
uint2 chi2(const uint2 a, const uint2 b, const uint2 c)
{ //keccak - chi
//	uint2 result = a ^ (~b) & c;
//	0xD2 ^ ((~0xCC) & 0xAA)
	uint2 result;
	asm("lop3.b32 %0, %1, %2, %3, 0xD2;" : "=r"(result.x) : "r"(a.x), "r"(b.x), "r"(c.x)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA
	asm("lop3.b32 %0, %1, %2, %3, 0xD2;" : "=r"(result.y) : "r"(a.y), "r"(b.y), "r"(c.y)); //0x96 = 0xF0 ^ 0xCC ^ 0xAA

	return result;
}


__global__	__launch_bounds__(1024,1)
void keccak256_gpu_hash_ZP(uint32_t threads, uint32_t *const __restrict__ resNounce, const uint2 highTarget)
{
	const uint2 keccak_round_constants35[24] = {
		{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
		{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
		{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
		{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
		{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
		{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
		{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
		{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
		{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
		{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
	};

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nounce = thread;
		uint2 bc[5], tmpxor[5], tmp1, tmp2;
		uint2 s[25];

		s[11] = c_PaddedMessageZP_PRE[5];
		uint2 k;
		k.x = cuda_swab32(nounce);
		k.y = 6;
		s[16].x = 0;
		s[16].y = 0x80000000;


		uint2 i = c_PaddedMessageZP_PRE[12] ^ k;
		bc[0] = c_PaddedMessageZP_PRE[2] ^ ROL2(i, 1);
		bc[1] = c_PaddedMessageZP_PRE[1];
		bc[2] = i ^ ROL2(c_PaddedMessageZP_PRE[3], 1);

		tmp1 = c_PaddedMessageZP_PRE[8] ^ bc[0];

		s[1] = ROL2(c_PaddedMessageZP_PRE[10] ^ bc[0], 44);
		s[2] = ROL2(k ^ bc[1], 43);
		s[12] = ROL2(bc[2], 25);
		s[19] = ROR8(bc[2]);
		s[24] = ROL2(bc[0], 2);
		s[21] = ROL2(c_PaddedMessageZP_PRE[19] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[5] = ROL2(c_PaddedMessageZP_PRE[17] ^ bc[2], 28);
		s[3] = ROL2( bc[2], 21);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[10] = ROL2(tmp1, 1);


		s[0] = c_PaddedMessageZP_PRE[0];
		s[4] = c_PaddedMessageZP_PRE[4];
		s[6] = c_PaddedMessageZP_PRE[6];
		s[7] = c_PaddedMessageZP_PRE[7];
		s[9] = c_PaddedMessageZP_PRE[9];
		s[11] = c_PaddedMessageZP_PRE[11];
		s[13] = c_PaddedMessageZP_PRE[13];
		s[14] = c_PaddedMessageZP_PRE[14];
		s[15] = c_PaddedMessageZP_PRE[15];
		s[16] = c_PaddedMessageZP_PRE[16];
		s[18] = c_PaddedMessageZP_PRE[18];
		s[20] = c_PaddedMessageZP_PRE[20];
		s[23] = c_PaddedMessageZP_PRE[21];
		s[22] = c_PaddedMessageZP_PRE[22];


		tmp1 = s[0];
		tmp2 = s[1];
		s[0] = chi2(s[0], s[1], s[2]);
		s[1] = chi2(s[1], s[2], s[3]);
		s[2] = chi2(s[2], s[3], s[4]);
		s[3] = chi2(s[3], s[4], tmp1);
		s[4] = chi2(s[4], tmp1, tmp2);


#pragma unroll
		for (int j = 5; j < 25; j += 5)
		{
			tmp1 = s[j];
			tmp2 = s[j + 1];
			s[j] = chi2(s[j], s[j + 1], s[j + 2]);
			s[j + 1] = chi2(s[j + 1], s[j + 2], s[j + 3]);
			s[j + 2] = chi2(s[j + 2], s[j + 3], s[j + 4]);
			s[j + 3] = chi2(s[j + 3], s[j + 4], tmp1);
			s[j + 4] = chi2(s[j + 4], tmp1, tmp2);
		}

		s[0].x ^= 1;

#pragma unroll
		for (int i = 1; i < 23; i++) 
		{

#pragma unroll
			for (int j = 0; j < 5; j++) {
				tmpxor[j] = vectorize(xor5(devectorize(s[j]), devectorize(s[j + 5]), devectorize(s[j + 10]), devectorize(s[j + 15]), devectorize(s[j + 20])));
			}

			for (int j = 0; j < 5; j++) {
				bc[j] = ROL2(tmpxor[j], 1);
			}
			s[4] = xor3x2(s[4], tmpxor[3], bc[0]); s[9] = xor3x2(s[9], tmpxor[3], bc[0]); s[14] = xor3x2(s[14], tmpxor[3], bc[0]); s[19] = xor3x2(s[19], tmpxor[3], bc[0]); s[24] = xor3x2(s[24], tmpxor[3], bc[0]);
			s[0] = xor3x2(s[0], tmpxor[4], bc[1]); s[5] = xor3x2(s[5], tmpxor[4], bc[1]); s[10] = xor3x2(s[10], tmpxor[4], bc[1]); s[15] = xor3x2(s[15], tmpxor[4], bc[1]); s[20] = xor3x2(s[20], tmpxor[4], bc[1]);
			s[1] = xor3x2(s[1], tmpxor[0], bc[2]); s[6] = xor3x2(s[6], tmpxor[0], bc[2]); s[11] = xor3x2(s[11], tmpxor[0], bc[2]); s[16] = xor3x2(s[16], tmpxor[0], bc[2]); s[21] = xor3x2(s[21], tmpxor[0], bc[2]);
			s[2] = xor3x2(s[2], tmpxor[1], bc[3]); s[7] = xor3x2(s[7], tmpxor[1], bc[3]); s[12] = xor3x2(s[12], tmpxor[1], bc[3]); s[17] = xor3x2(s[17], tmpxor[1], bc[3]); s[22] = xor3x2(s[22], tmpxor[1], bc[3]);
			s[3] = xor3x2(s[3], tmpxor[2], bc[4]); s[8] = xor3x2(s[8], tmpxor[2], bc[4]); s[13] = xor3x2(s[13], tmpxor[2], bc[4]); s[18] = xor3x2(s[18], tmpxor[2], bc[4]); s[23] = xor3x2(s[23], tmpxor[2], bc[4]);

			tmp1 = s[1];
			s[1] = ROL2(s[6], 44);	s[6] = ROL2(s[9], 20);	s[9] = ROL2(s[22], 61);	s[22] = ROL2(s[14], 39);
			s[14] = ROL2(s[20], 18);	s[20] = ROL2(s[2], 62);	s[2] = ROL2(s[12], 43);	s[12] = ROL2(s[13], 25);
			s[13] = ROL8(s[19]);	s[19] = ROR8(s[23]);	s[23] = ROL2(s[15], 41);	s[15] = ROL2(s[4], 27);
			s[4] = ROL2(s[24], 14);	s[24] = ROL2(s[21], 2);	s[21] = ROL2(s[8], 55);	s[8] = ROL2(s[16], 45);
			s[16] = ROL2(s[5], 36);	s[5] = ROL2(s[3], 28);	s[3] = ROL2(s[18], 21);	s[18] = ROL2(s[17], 15);
			s[17] = ROL2(s[11], 10);	s[11] = ROL2(s[7], 6);	s[7] = ROL2(s[10], 3);	s[10] = ROL2(tmp1, 1);

			#pragma unroll
			for (int j = 0; j < 25; j += 5)
			{
				tmp1 = s[j];
				tmp2 = s[j + 1];
				s[j] = chi2(s[j], s[j + 1], s[j + 2]);
				s[j + 1] = chi2(s[j + 1], s[j + 2], s[j + 3]);
				s[j + 2] = chi2(s[j + 2], s[j + 3], s[j + 4]);
				s[j + 3] = chi2(s[j + 3], s[j + 4], tmp1);
				s[j + 4] = chi2(s[j + 4], tmp1, tmp2);
			}
			s[0].x ^= keccak_round_constants35[i].x;
			s[0].y ^= keccak_round_constants35[i].y;
		}

#pragma unroll
		for (int j = 0; j < 5; j++) {
			tmpxor[j] = vectorize(xor5(devectorize(s[j]), devectorize(s[j + 5]), devectorize(s[j + 10]), devectorize(s[j + 15]), devectorize(s[j + 20])));
		}

		for (int j = 0; j < 5; j++) {
			bc[j] = ROL2(tmpxor[j], 1);
		}
		s[4] = xor3x2(s[4], tmpxor[3], bc[0]); s[9] = xor3x2(s[9], tmpxor[3], bc[0]); s[14] = xor3x2(s[14], tmpxor[3], bc[0]); s[19] = xor3x2(s[19], tmpxor[3], bc[0]); s[24] = xor3x2(s[24], tmpxor[3], bc[0]);
		s[0] = xor3x2(s[0], tmpxor[4], bc[1]); s[5] = xor3x2(s[5], tmpxor[4], bc[1]); s[10] = xor3x2(s[10], tmpxor[4], bc[1]); s[15] = xor3x2(s[15], tmpxor[4], bc[1]); s[20] = xor3x2(s[20], tmpxor[4], bc[1]);
		s[1] = xor3x2(s[1], tmpxor[0], bc[2]); s[6] = xor3x2(s[6], tmpxor[0], bc[2]); s[11] = xor3x2(s[11], tmpxor[0], bc[2]); s[16] = xor3x2(s[16], tmpxor[0], bc[2]); s[21] = xor3x2(s[21], tmpxor[0], bc[2]);
		s[2] = xor3x2(s[2], tmpxor[1], bc[3]); s[7] = xor3x2(s[7], tmpxor[1], bc[3]); s[12] = xor3x2(s[12], tmpxor[1], bc[3]); s[17] = xor3x2(s[17], tmpxor[1], bc[3]); s[22] = xor3x2(s[22], tmpxor[1], bc[3]);
		s[3] = xor3x2(s[3], tmpxor[2], bc[4]); s[8] = xor3x2(s[8], tmpxor[2], bc[4]); s[13] = xor3x2(s[13], tmpxor[2], bc[4]); s[18] = xor3x2(s[18], tmpxor[2], bc[4]); s[23] = xor3x2(s[23], tmpxor[2], bc[4]);

		tmp1 = s[1];
		s[1] = ROL2(s[6], 44);	s[6] = ROL2(s[9], 20);	s[9] = ROL2(s[22], 61);	s[22] = ROL2(s[14], 39);
		s[14] = ROL2(s[20], 18);	s[20] = ROL2(s[2], 62);	s[2] = ROL2(s[12], 43);	s[12] = ROL2(s[13], 25);
		s[13] = ROL8(s[19]);	s[19] = ROR8(s[23]);	s[23] = ROL2(s[15], 41);	s[15] = ROL2(s[4], 27);
		s[4] = ROL2(s[24], 14);	s[24] = ROL2(s[21], 2);	s[21] = ROL2(s[8], 55);	s[8] = ROL2(s[16], 45);
		s[16] = ROL2(s[5], 36);	s[5] = ROL2(s[3], 28);	s[3] = ROL2(s[18], 21);	s[18] = ROL2(s[17], 15);
		s[17] = ROL2(s[11], 10);	s[11] = ROL2(s[7], 6);	s[7] = ROL2(s[10], 3);	s[10] = ROL2(tmp1, 1);

#pragma unroll
		for (int j = 0; j < 25; j += 5)
		{
			tmp1 = s[j];
			tmp2 = s[j + 1];
			s[j] = chi2(s[j], s[j + 1], s[j + 2]);
			s[j + 1] = chi2(s[j + 1], s[j + 2], s[j + 3]);
			s[j + 2] = chi2(s[j + 2], s[j + 3], s[j + 4]);
			s[j + 3] = chi2(s[j + 3], s[j + 4], tmp1);
			s[j + 4] = chi2(s[j + 4], tmp1, tmp2);
		}
		s[0].x ^= keccak_round_constants35[23].x;
		s[0].y ^= keccak_round_constants35[23].y;

		if (cuda_swab32(s[0].x) <= (highTarget.x) && ( cuda_swab32(s[0].y) <= (highTarget.y)) )
		{
			const uint32_t tmp = atomicExch(&resNounce[0], nounce);
			if (tmp != UINT32_MAX)
				resNounce[1] = tmp;
		}
	}
}

__host__
void keccak256_cpu_hash_ZP(int thr_id, uint32_t threads, uint32_t *h_nounce, uint2 hightarget)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_KNonce[thr_id], 0xff, 2 * sizeof(uint32_t), gpustream[thr_id]));
	const uint32_t threadsperblock = 1024;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);
	keccak256_gpu_hash_ZP << <grid, block, 0, gpustream[thr_id] >> >(threads, d_KNonce[thr_id], hightarget);
//	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(h_nounce, d_KNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

__host__
void keccak256_setBlock_ZP(int thr_id, void *pdata)
{
	unsigned char PaddedMessage[104];
	memcpy(PaddedMessage, pdata, 104);
	

	uint64_t* PaddedMessageZP = (uint64_t*)PaddedMessage;
	uint64_t bc[5], tmpxor[5];
	uint64_t s[24];

	s[11] = PaddedMessageZP[11];
	s[12] = PaddedMessageZP[2] ^ PaddedMessageZP[7];			//make_uint2(cuda_swab32(nounce), 0x06);
	s[16] = (uint64_t)1 << 63;//(uint64_t)make_uint2(0, 0x80000000);

	s[2] = tmpxor[0] = PaddedMessageZP[0] ^ PaddedMessageZP[5] ^ PaddedMessageZP[10];
	tmpxor[1] = PaddedMessageZP[1] ^ PaddedMessageZP[6] ^ s[11] ^ s[16];
	tmpxor[3] = PaddedMessageZP[3] ^ PaddedMessageZP[8];
	s[3] = tmpxor[4] = PaddedMessageZP[4] ^ PaddedMessageZP[9];

	bc[1] = tmpxor[1] ^ ROTL64(tmpxor[3], 1);
	bc[3] = tmpxor[3] ^ ROTL64(tmpxor[0], 1);
	bc[4] = tmpxor[4] ^ ROTL64(tmpxor[1], 1);

//	tmp1 = c_PaddedMessageZP[1] ^ bc[0];
	s[1] = bc[1];


	s[0] = PaddedMessageZP[0] ^ bc[4];
//	s[1] = ROL2(c_PaddedMessageZP[6] ^ bc[0], 44);
	s[6] = ROTL64(PaddedMessageZP[9] ^ bc[3], 20);
	s[9] = ROTL64(bc[1], 61);
	s[22] = ROTL64(bc[3], 39);
	s[14] = ROTL64(bc[4], 18);
	s[20] = ROTL64(PaddedMessageZP[2] ^ bc[1], 62);
//	s[2] = ROTL64(s[12] ^ bc[1], 43);
//	s[12] = ROTL64(bc[2], 25);
	s[13] = ROTL64(bc[3], 8);
//	s[19] = ROR8(bc[2]);
	s[21] = ROTL64(bc[4], 41);
	s[15] = ROTL64(PaddedMessageZP[4] ^ bc[3], 27);
	s[4] = ROTL64(bc[3], 14);
//	s[24] = ROL2(bc[0], 2);
//	s[21] = ROL2(PaddedMessageZP[8] ^ bc[2], 55);
//	s[8] = ROL2(s[16] ^ bc[0], 45);
	s[16] = ROTL64(PaddedMessageZP[5] ^ bc[4], 36);
//	s[5] = ROL2(c_PaddedMessageZP[3] ^ bc[2], 28);
//	s[3] = ROL2(bc[2], 21);
	s[18] = ROTL64(bc[1], 15);
//	s[17] = ROL2(s[11] ^ bc[0], 10);
	s[11] = ROTL64(PaddedMessageZP[7] ^ bc[1], 6);
	s[7] = ROTL64(PaddedMessageZP[10] ^ bc[4], 3);
//	s[10] = ROL2(tmp1, 1);

	s[5] = PaddedMessageZP[11];
	s[8] = PaddedMessageZP[1];
	s[10] = PaddedMessageZP[6];
	s[19] = PaddedMessageZP[8];
	s[17] = PaddedMessageZP[3];
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_PaddedMessageZP_PRE), &s[0], 23 * sizeof(uint64_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));

//	if (opt_debug)
//		CUDA_SAFE_CALL(hipDeviceSynchronize());
}


__host__
void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_nounce)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_KNonce[thr_id], 0xff, 2 * sizeof(uint32_t), gpustream[thr_id]));
	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	keccak256_gpu_hash_80<<<grid, block, 0, gpustream[thr_id]>>>(threads, startNounce, d_KNonce[thr_id]);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(h_nounce, d_KNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}



__global__ __launch_bounds__(256,3)
void keccak256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		uint2 keccak_gpu_state[25];
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4) keccak_gpu_state[i] = vectorize(outputHash[i*threads+thread]);
			else     keccak_gpu_state[i] = UINT2(0, 0);
		}
		keccak_gpu_state[4]  = UINT2(0x06, 0);
		keccak_gpu_state[16] = UINT2(0, 0x80000000);
		keccak_block(keccak_gpu_state);

		#pragma unroll 4
		for (int i=0; i<4; i++)
			outputHash[i*threads+thread] = devectorize(keccak_gpu_state[i]);
	}
}

__host__
void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	keccak256_gpu_hash_32 <<<grid, block, 0, gpustream[thr_id]>>> (threads, startNounce, d_outputHash);
	CUDA_SAFE_CALL(hipGetLastError());
}

__host__
void keccak256_setBlock_80(int thr_id, void *pdata,const void *pTargetIn)
{
	unsigned char PaddedMessage[80];
	memcpy(PaddedMessage, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 10 * sizeof(uint64_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__host__
void keccak256_cpu_init(int thr_id, uint32_t threads)
{
	CUDA_SAFE_CALL(hipMalloc(&d_KNonce[thr_id], 2*sizeof(uint32_t)));
}